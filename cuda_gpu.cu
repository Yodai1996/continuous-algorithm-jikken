#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#define M 32 

__global__
void Calcu(float **od, float **ev)
{
  int i=blockIdx.x * blockDim.x + threadIdx.x;
  int j=blockIdx.y * blockDim.y + threadIdx.y;
        int count;
        float r=0.2;

        if(i<M && j<M){
        for(count=0;count<100;count+=2){
                od[i][j]=(1-4*r)*ev[i][j]+r*ev[i+1][j]+r*ev[i-1][j]+r*ev[i][j+1]+r*ev[i][j-1];
                __syncthreads();

                ev[i][j]=(1-4*r)*od[i][j]+r*od[i+1][j]+r*od[i-1][j]+r*od[i][j+1]+r*od[i][j-1];
                __syncthreads();
        }
}}


int main(void)
{
        struct timeval t0,t1;
        int i,j;
        float dx=1/M;
        float r=0.2;
        float even[M+1][M+1], odd[M+1][M+1];
        float time;
        size_t pitch;

        float **ev;
        float **od;

        for(i=0;i<M+1;i++)
                for(j=0;j<M+1;j++)
                        even[i][j]=1;

        for(i=0;i<M+1;i++){
                even[0][i]=0;
                even[M][i]=0;
                even[i][0]=0;
                even[i][M]=0;
        }
        for(i=0;i<M+1;i++){
                odd[0][i]=0;
                odd[M][i]=0;
                odd[i][0]=0;
                odd[i][M]=0;
        }

        hipMallocPitch(ev, &pitch, (M+1)*sizeof(float), M+1);
        hipMallocPitch(od, &pitch, (M+1)*sizeof(float), M+1);
   
        hipMemcpy2D(ev ,pitch ,even ,pitch, (M+1)*sizeof(float), M+1, hipMemcpyHostToDevice);
        hipMemcpy2D(od ,pitch ,odd  ,pitch, (M+1)*sizeof(float), M+1, hipMemcpyHostToDevice);

        dim3 threadPerBlock(32,32);
        dim3 numBlocks(M/threadPerBlock.x, M/threadPerBlock.y);

        gettimeofday(&t0, NULL);

        Calcu<<<numBlocks, threadPerBlock>>>(od , ev);

        hipDeviceSynchronize();

        hipMemcpy2D(even ,pitch ,ev ,pitch, (M+1)*sizeof(float), M+1, hipMemcpyDeviceToHost);
        hipMemcpy2D(odd  ,pitch ,od ,pitch, (M+1)*sizeof(float), M+1, hipMemcpyDeviceToHost);

        gettimeofday(&t1, NULL);

        time = t1.tv_sec-t0.tv_sec + (t1.tv_usec - t0.tv_usec)*1.0e-6;
        printf("Elapsed time = %lf\n", time);
        printf("FLOPS = %lf\n", (M-1)*(M-1)*100*6/time);


        for(j=10;j>=0;j--){
                for(i=0;i<11;i++)
                        printf("%f ",even[i][j]);
                printf("\n");
        }

        return 0;
}
