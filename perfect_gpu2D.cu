#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#define M 1024 

__global__
void Calcu(float *address, size_t pitch)
{
  //pitchの使い道がわからん。詰んだ。
  int i=blockIdx.x * blockDim.x + threadIdx.x;
  int j=blockIdx.y * blockDim.y + threadIdx.y;
  float up, down, left, right;
  
  float r=0.2;
  
  for(int count=0; count<100; count++){
    if(0<i && i<M && 0<j && j<M){
      float* row=(float*)((char*)address+j*pitch);
      float* up_row=(float*)((char*)address+(j-1)*pitch);
      float* down_row=(float*)((char*)address+(j+1)*pitch);
      up=up_row[i];
      down=down_row[i];
      left=row[i-1];
      right=row[i+1];
      __syncthreads();
      row[i]=(1-4*r)*row[i]+r*up+r*down+r*left+r*right;
      }
      __syncthreads();
  }
}


int main(void)
{
        struct timeval t0,t1;
        int i,j;
        float dx=1/M;
        float r=0.2;
        float array[M+1][M+1];
        float time;
        size_t pitch;

        float *address;

        for(i=0;i<M+1;i++)
                for(j=0;j<M+1;j++)
                        array[i][j]=1;

        for(i=0;i<M+1;i++){
                array[0][i]=0;
                array[M][i]=0;
                array[i][0]=0;
                array[i][M]=0;
        }

        hipMallocPitch(&address, &pitch, (M+1)*sizeof(float), M+1);
   
        hipMemcpy2D(address ,pitch ,array ,(M+1)*sizeof(float), (M+1)*sizeof(float), M+1, hipMemcpyHostToDevice);

        dim3 threadPerBlock(32,32);
        dim3 numBlock(M/threadPerBlock.x, M/threadPerBlock.y);

        gettimeofday(&t0, NULL);

        Calcu<<<numBlock, threadPerBlock>>>(address, pitch);

        hipDeviceSynchronize();

        hipMemcpy2D(array ,(M+1)*sizeof(float) ,address ,pitch, (M+1)*sizeof(float), M+1, hipMemcpyDeviceToHost);
        hipFree(address);

        gettimeofday(&t1, NULL);

        time = t1.tv_sec-t0.tv_sec + (t1.tv_usec - t0.tv_usec)*1.0e-6;
        printf("Elapsed time = %lf\n", time);
        printf("FLOPS = %lf\n", (M-1)*(M-1)*100*6/time);


        for(j=10;j>=0;j--){
                for(i=0;i<11;i++)
                        printf("%f ",array[i][j]);
                printf("\n");
        }

        return 0;
}
